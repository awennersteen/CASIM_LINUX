#include "hip/hip_runtime.h"

#ifndef __MPI_BEHAVIOR_
#define __MPI_BEHAVIOR_

#include <stdio.h>
#include "cMacros.h"


#ifdef MPI_PATHS

__device__ int d_behavior(unsigned int x, unsigned int y, float4 *agent, float4 *ids, int *d_world, int world_width, int world_height, int agent_width)
{
	//HPV movimiento de los agentes
	// 1ra opcion seguir en la misma direccion, se revisa el mundo para ver si esta disponible las sig coordenadas
	// en caso contrario se mueve 45 grados en sentido antihorario

	float PI = 3.14159265358979323846;
	int rebote = 0;
	int limit_width = world_width/40;
	int limit_height = world_height/40;

	//Verificando si estas cerca de la orilla en la siguiente coordenada
	//next position = actual position + (cos(teta)*velocity)
	int ccx = round( agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z) * 2 * agent[y*agent_width+x].w) );
	int ccy = round( agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z) * 2 * agent[y*agent_width+x].w) );
	if( ccx < limit_width || ccx > world_width - limit_width || ccy < limit_height || ccy > world_height - limit_height )
	{
		//si la siguiente coordenada sale del mundo entonces el angulo cambia 90 grados "rebote"
		agent[y*agent_width+x].z += PI/2;
		rebote = 1;
	}

	//calculando las coordenadas originales
	int cx_old = round( agent[y*agent_width+x].x ) ;
	int cy_old = ( round( agent[y*agent_width+x].y ) - 1 ) * world_width ;
	if ( cy_old < 0 )
		cy_old = 0 ;
	int coord_old =  cy_old + cx_old ;

	//Aqui revisas que la nueva posicion no este ocupada, si se trata de un rebote haces una excepcion y permites la colision
	// si esta ocupada la sig posicion te vas moviendo 45 grados en sentido antihorario
	// se utiliza world_width porque se hace la conversion a un arreglo dimensional de uno bidimensional
	int cx = round( agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z) * agent[y*agent_width+x].w) );
	int cy = (  round( agent[y*agent_width+x].y +  (sin(agent[y*agent_width+x].z)  * agent[y*agent_width+x].w) ) - 1  )* world_width ;
	int coord =  cx + cy ;
	if( d_world[coord] == 0 || rebote )
	{
		//cambias el valor a las siguientes posiciones
		agent[y*agent_width+x].x = agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z) * agent[y*agent_width+x].w) ;
		agent[y*agent_width+x].y = agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z) * agent[y*agent_width+x].w) ;
		//pones el id del agente en la casilla del mundo que esta ocupando
		d_world[coord] = ids[y*agent_width+x].x ;
		//liberas la casilla ocupada anteriormente
		d_world[coord_old] = 0;
	}
	else{
		cx = round( agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + PI/4)  *  agent[y*agent_width+x].w ) ) ;
		cy = (  round( agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + PI/4)  * agent[y*agent_width+x].w) ) -1  ) * world_width ;
		coord =  cy + cx ;
		if( d_world[coord] == 0)
		{
			agent[y*agent_width+x].x = agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + PI/4) *agent[y*agent_width+x].w) ;
			agent[y*agent_width+x].y = agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + PI/4) *agent[y*agent_width+x].w) ;
			d_world[coord] = ids[y*agent_width+x].x ;
			d_world[coord_old] = 0;
		}
		else{
			cx = round( agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + 2*PI/4)  * agent[y*agent_width+x].w) ) ;
			cy = ( round( agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + 2*PI/4)  * agent[y*agent_width+x].w) ) - 1) * world_width;
			coord =  cy + cx ;
			if( d_world[coord] == 0)
			{
				agent[y*agent_width+x].x = agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + 2*PI/4) *agent[y*agent_width+x].w) ;
				agent[y*agent_width+x].y = agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + 2*PI/4) *agent[y*agent_width+x].w) ;
				d_world[coord] = ids[y*agent_width+x].x ;
				d_world[coord_old] = 0;
			}
			else{
				cx = round( agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + 3*PI/4)  * agent[y*agent_width+x].w) ) ;
				cy = ( round( agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + 3*PI/4) * agent[y*agent_width+x].w) ) -1 ) * world_width ;
				coord =  cy + cx ;
				if( d_world[coord] == 0)
				{
					agent[y*agent_width+x].x = agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + 3*PI/4) *agent[y*agent_width+x].w) ;
					agent[y*agent_width+x].y = agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + 3*PI/4) *agent[y*agent_width+x].w) ;
					d_world[coord] = ids[y*agent_width+x].x ;
					d_world[coord_old] = 0;
				}
				else{
					cx = round( agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + PI)  * agent[y*agent_width+x].w) ) ;
					cy = ( round( agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + PI) * agent[y*agent_width+x].w) ) -1 ) * world_width;
					coord =  cy + cx ;
					if( d_world[coord] == 0)
					{
						agent[y*agent_width+x].x = agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + PI) *agent[y*agent_width+x].w) ;
						agent[y*agent_width+x].y = agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + PI) *agent[y*agent_width+x].w) ;
						d_world[coord] = ids[y*agent_width+x].x ;
						d_world[coord_old] = 0;
					}
					else{
						cx = round( agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + 5*PI/4)  * agent[y*agent_width+x].w) ) ;
						cy = ( round( agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + 5*PI/4)  * agent[y*agent_width+x].w) ) - 1 ) * world_width;
						coord =  cy + cx ;
						if( d_world[coord] == 0)
						{
							agent[y*agent_width+x].x = agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + 5*PI/4) *agent[y*agent_width+x].w) ;
							agent[y*agent_width+x].y = agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + 5*PI/4) *agent[y*agent_width+x].w) ;
							d_world[coord] = ids[y*agent_width+x].x ;
							d_world[coord_old] = 0;
						}
						else{
							cx = round( agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + 6*PI/4) * agent[y*agent_width+x].w) ) ;
							cy = ( round( agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + 6*PI/4) * agent[y*agent_width+x].w) ) -1 ) * world_width;
							coord =  cy + cx ;
							if( d_world[coord] == 0)
							{
								agent[y*agent_width+x].x = agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + 6*PI/4) *agent[y*agent_width+x].w) ;
								agent[y*agent_width+x].y = agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + 6*PI/4) *agent[y*agent_width+x].w) ;
								d_world[coord] = ids[y*agent_width+x].x ;
								d_world[coord_old] = 0;
							}
							else{
								cx = round( agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + 7*PI/4) * agent[y*agent_width+x].w) ) ;
								cy = ( round( agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + 7*PI/4) * agent[y*agent_width+x].w) ) - 1 ) * world_width;
								coord =  cy + cx ;
								if( d_world[coord] == 0)
								{
									agent[y*agent_width+x].x = agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + 7*PI/4) *agent[y*agent_width+x].w) ;
									agent[y*agent_width+x].y = agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + 7*PI/4) *agent[y*agent_width+x].w) ;
									d_world[coord] = ids[y*agent_width+x].x ;
									d_world[coord_old] = 0;
								}
								else{
									//si todas las posiciones a su alrededor estan ocupadas se queda donde esta y marcas
									//ocupada de nuevo esa posicion
									//d_world[coord_old] = 1;
	//si todas las posiciones a su alrededor estan ocupadas avanzas en la direccion original aunque se colisione
	cx = round( agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z) * agent[y*agent_width+x].w) ) ;
	cy = ( round( agent[y*agent_width+x].y +  ( sin(agent[y*agent_width+x].z)  * agent[y*agent_width+x].w) ) - 1 )* world_width ;
	coord =  cx + cy ;

	agent[y*agent_width+x].x = agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z) *agent[y*agent_width+x].w)*32.0 ;
	agent[y*agent_width+x].y = agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z) *agent[y*agent_width+x].w)*32.0 ;
	d_world[coord] = ids[y*agent_width+x].x ;
	d_world[coord_old] = 0;
								}
							} //7*PI/4
						} //6*PI/4
					}//5*PI/4
				}//PI
			}//3*PI/4
		 }//PI/2
	}//PI/4

	return coord;

}

__device__ void d_interchange(unsigned int x, unsigned int y, float4 *agent, float4 *ids, int *d_world, int agent_width, int world_height_node,
		int world_width_node, int num_comp_nodes, int coord)
{
   int lookup_node4[2][2];
   lookup_node4[0][0] = 1;
   lookup_node4[0][1] = 2;
   lookup_node4[1][0] = 3;
   lookup_node4[1][1] = 4;

   int lookup_node16[4][4];
   lookup_node16[0][0] = 1;
   lookup_node16[0][1] = 2;
   lookup_node16[0][2] = 3;
   lookup_node16[0][3] = 4;
   lookup_node16[1][0] = 5;
   lookup_node16[1][1] = 6;
   lookup_node16[1][2] = 7;
   lookup_node16[1][3] = 8;
   lookup_node16[2][0] = 9;
   lookup_node16[2][1] = 10;
   lookup_node16[2][2] = 11;
   lookup_node16[2][3] = 12;
   lookup_node16[3][0] = 13;
   lookup_node16[3][1] = 14;
   lookup_node16[3][2] = 15;
   lookup_node16[3][3] = 16;

   int lookup_node9[3][3];
   lookup_node9[0][0] = 1;
   lookup_node9[0][1] = 2;
   lookup_node9[0][2] = 3;
   lookup_node9[1][0] = 4;
   lookup_node9[1][1] = 5;
   lookup_node9[1][2] = 6;
   lookup_node9[2][0] = 7;
   lookup_node9[2][1] = 8;
   lookup_node9[2][2] = 9;

	if (num_comp_nodes == 4)
	{
		ids[y*agent_width+x].z = lookup_node4[(int)agent[y*agent_width+x].y/world_height_node][(int)agent[y*agent_width+x].x/world_width_node];
		d_world[coord] = 0 ;
	}
	else if (num_comp_nodes == 16)
	{
		ids[y*agent_width+x].z = lookup_node16[(int)agent[y*agent_width+x].y/world_height_node][(int)agent[y*agent_width+x].x/world_width_node];
		d_world[coord] = 0 ;
	}
	else if (num_comp_nodes == 9)
	{
		ids[y*agent_width+x].z = lookup_node9[(int)agent[y*agent_width+x].y/world_height_node][(int)agent[y*agent_width+x].x/world_width_node];
		d_world[coord] = 0 ;
	}

}

__device__ bool d_border(int x, int y, float4 *agent, int world_width, int agent_width, int world_height_node,
		int world_width_node, int pid, int num_comp_nodes)
{
	int border_width = 5;
	int nodes_per_row = sqrtf(num_comp_nodes);
	int x_limit_right = (( ( (pid - 1) % nodes_per_row ) + 1 ) * world_width_node);
	int x_limit_left = (( (pid - 1) % nodes_per_row )  * world_width_node);
	int num_row = (pid - 1) / nodes_per_row;
	int y_limit_up = (( num_row + 1 ) * world_height_node);
	int y_limit_down = (num_row  * world_height_node);

	//getting coords
	int cx = round( agent[y*agent_width+x].x ) ;
	int cy = round( agent[y*agent_width+x].y ) ;


	if( cx < x_limit_left + border_width )
	{
		return true;
	}

	if( cx > x_limit_right - border_width )
	{
		return true;
	}

	if( cy < y_limit_down + border_width )
	{
		return true;
	}

	if( cy > y_limit_up - border_width )
	{
		return true;
	}

	return false;
}

///////////////////////////////////////////////////////////////////////////////
//! Kernel to modify internal agents positions
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void kernel_internal(float4 *agent, float4 *ids, int *d_world, int world_width, int world_height, int world_height_node,
		int world_width_node, int pid, int num_comp_nodes, int count_agents_total, int grid_width)
{
   unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
   unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
   int coord;
   int agent_width = grid_width;

   if ( y * agent_width + x < count_agents_total ) //verify bounds
   {
	   if(ids[y*agent_width+x].z == pid) //check if agent is active
	   {
		   if(!d_border(x, y, agent, world_width, agent_width, world_height_node, world_width_node, pid, num_comp_nodes)) //check is not in the border
		   {
			   ids[y*agent_width+x].w += 1; //iterations counter

			   coord = d_behavior(x, y, agent, ids, d_world, world_width, world_height, agent_width);
			   d_interchange(x, y, agent, ids, d_world, agent_width, world_height_node, world_width_node, num_comp_nodes, coord);
		   }
	   }
	   else
		   ids[y*agent_width+x].z = -1;
   }
}

///////////////////////////////////////////////////////////////////////////////
//! Kernel to modify border agents positions
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void kernel_border(float4 *agent, float4 *ids, int *d_world, int world_width, int world_height, int world_height_node,
		int world_width_node, int pid, int num_comp_nodes, int count_agents_total, float4 *agent_fr, float4 *ids_fr, int grid_width)
{
   unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
   unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
   int agent_width = grid_width;
   int coord;


   if ( y * agent_width + x < count_agents_total ) //verify bounds of the array
   {
	   if(ids[y*agent_width+x].z == pid) //check if agent is active
	   {
		   if(d_border(x, y, agent, world_width, agent_width, world_height_node, world_width_node, pid, num_comp_nodes)) //check is in the border
		   {

			   ids[y*agent_width+x].w += 1; //iterations counter

			   coord = d_behavior(x, y, agent, ids, d_world, world_width, world_height, agent_width);
			   d_interchange(x, y, agent, ids, d_world, agent_width, world_height_node, world_width_node, num_comp_nodes, coord);
			   agent_fr[y*agent_width+x] = agent[y*agent_width+x];
			   ids_fr[y*agent_width+x] = ids[y*agent_width+x];
		   }
	   }
	   else
		   ids[y*agent_width+x].z = -1;
   }
}

// CUDA computation on each node
// No MPI here, only CUDA
extern "C" void launch_kernel_border(float4 *d_agents_in, float4 *d_agents_ids, int *d_world, int world_width, int world_height, int world_height_node,
		int world_width_node, int pid, int num_comp_nodes, int count_agents_total, float4 *d_agents_in_fr, float4 *d_agents_ids_fr,	hipStream_t stream0)
{
    // execute the kernel
    int block_width = 8;
    int block_height = 8;
    dim3 block(block_width, block_height, 1);
    int agent_width = sqrt(count_agents_total);
    dim3 grid((agent_width / block.x) + 1, (agent_width / block.y) + 1, 1);

    kernel_border<<< grid, block, 0, stream0>>>(d_agents_in, d_agents_ids, d_world, world_width, world_height, world_height_node, world_width_node,
    		pid, num_comp_nodes, count_agents_total, d_agents_in_fr, d_agents_ids_fr, agent_width);
}

extern "C" void launch_kernel_internal(float4 *d_agents_in, float4 *d_agents_ids, int *d_world, int world_width, int world_height,
		int world_height_node, int world_width_node, int pid, int num_comp_nodes, int count_agents_total, hipStream_t stream1 )
{
    // execute the kernel
    int block_width = 8;
    int block_height = 8;
    dim3 block(block_width, block_height, 1);
    int agent_width = sqrt(count_agents_total);
    dim3 grid((agent_width / block.x) + 1, (agent_width / block.y) + 1, 1);

//printf ("\n\n BEGIN: launch_kernel_internal() \n");

    kernel_internal<<< grid, block>>>(d_agents_in, d_agents_ids, d_world, world_width, world_height, world_height_node, world_width_node,
    		pid, num_comp_nodes, count_agents_total, agent_width);

//printf ("\n\n END: launch_kernel_internal() \n");
}


#endif
#endif

#ifdef MPI_PATHS_ON_NODES

#define KERNEL

#ifdef KERNEL
//Kernel 2
__device__ int d_behavior(unsigned int x, unsigned int y, float4 *d_agents, float4 *d_ids, int *d_world, int world_width, int world_height, int agent_width)
{
	float PI = 3.14159265358979323846;
	int limit_width = world_width/40;
	int limit_height = world_height/40;
	int radio = 5;
	int number_directions = 8;
    //int points[number_directions]
	int points[8];
	//int nl[number_directions*radio]
	int nl[ 40 ];
	int i, j, direction, k=0;
	int coord;
	float cx, cy, ai;
	float cx_new, cy_new;
	int coord_new, p=0, mx;

    float angle = d_agents[y*agent_width+x].z;
    int ax = d_agents[y*agent_width+x].x;
    int ay = d_agents[y*agent_width+x].y;
    int aw = d_agents[y*agent_width+x].w;
    float cos_angle = cosf(angle);
    float sin_angle = sinf(angle);



    //if next cell is out of the world the angle change 90 degrees
	int ccx = ax + ( cos_angle * 2 * aw) ;
	int ccy = ay + ( sin_angle * 2 * aw) ;
	if( (ccx < limit_width || ccx > world_width - limit_width) || (ccy < limit_height || ccy > world_height - limit_height) )
	{
		angle += PI/2;
		//d_agents[y*agent_width+x].z *= -1;
	}

	//get the original coord to make available
	float cx_old = ax;
	float cy_old = ( ay - 1 ) * world_width ;
	if ( cy_old < 0 )
		cy_old = 0 ;
	int coord_old =  cy_old + cx_old;


	//Listing cells in the neighborhood (raytrace over a radio)
	ai = angle;
	#pragma unroll
	for(k=0; k < radio*number_directions;  )
	{
		ai += PI/4;
		for(j = 1; j <= radio; j++)
		{
			cx = cx_old + ( cosf(ai) * j);
			cy = cy_old + ( sinf(ai) * j);
			coord = round(cx + cy);
			nl[k] = d_world[coord];
			k++;
		}
	}


	//Evaluating the path
	for(direction = 0; direction < number_directions; direction++)
	{
		p = 0;
		for(i=0; i < radio; i++)
		{
			if(nl[i + (direction*radio)] == 0)
				p++;
			else if( round(d_agents[ nl[i + (direction*radio)] - 1 ].z) == round(angle) )
				p++;
			else
				break;
		}
		points[direction] = p;
	}

	//Choosing the best path
	mx = points[0];
	direction = 0;
	for(i=1;i<number_directions;i++)
	{
		if(mx < points[i])
		{
			mx = points[i];
			direction = i;
		}
	}

	//Updating values
	angle += direction * PI/4;
	//Asign new coords
	d_agents[y*agent_width+x].x = round( ax + ( cosf(angle) * aw ) ) ;
	d_agents[y*agent_width+x].y = round( ay + ( sinf(angle) * aw ) ) ;
	d_agents[y*agent_width+x].z = angle;
	cx_new = d_agents[y*agent_width+x].x;
	cy_new = ( d_agents[y*agent_width+x].y - 1 ) * world_width ;
	if ( cy_new < 0 )
		cy_new = 0 ;
	coord_new =  cx_new + cy_new ;
	//set as busy cell

	d_world[coord_new] = d_ids[y*agent_width+x].x ;
	//release old cell
	d_world[coord_old] = 0;

	return coord_new;
}
#else


__device__ int d_behavior(unsigned int x, unsigned int y, float4 *agent, float4 *ids, int *d_world, int world_width, int world_height, int agent_width)
{
	//HPV movimiento de los agentes
	// 1ra opcion seguir en la misma direccion, se revisa el mundo para ver si esta disponible las sig coordenadas
	// en caso contrario se mueve 45 grados en sentido antihorario

	float PI = 3.14159265358979323846;
	int rebote = 0;
	int limit_width = world_width/40;
	int limit_height = world_height/40;

	//Verificando si estas cerca de la orilla en la siguiente coordenada
	//next position = actual position + (cos(teta)*velocity)
	int ccx = round( agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z) * 2 * agent[y*agent_width+x].w) );
	int ccy = round( agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z) * 2 * agent[y*agent_width+x].w) );
	if( ccx < limit_width || ccx > world_width - limit_width || ccy < limit_height || ccy > world_height - limit_height )
	{
		//si la siguiente coordenada sale del mundo entonces el angulo cambia 90 grados "rebote"
		agent[y*agent_width+x].z += PI/2;
		rebote = 1;
	}

	//calculando las coordenadas originales
	int cx_old = round( agent[y*agent_width+x].x ) ;
	int cy_old = ( round( agent[y*agent_width+x].y ) - 1 ) * world_width ;
	if ( cy_old < 0 )
		cy_old = 0 ;
	int coord_old =  cy_old + cx_old ;

	//Aqui revisas que la nueva posicion no este ocupada, si se trata de un rebote haces una excepcion y permites la colision
	// si esta ocupada la sig posicion te vas moviendo 45 grados en sentido antihorario
	// se utiliza world_width porque se hace la conversion a un arreglo dimensional de uno bidimensional
	int cx = round( agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z) * agent[y*agent_width+x].w) );
	int cy = (  round( agent[y*agent_width+x].y +  (sin(agent[y*agent_width+x].z)  * agent[y*agent_width+x].w) ) - 1  )* world_width ;
	int coord =  cx + cy ;
	if( d_world[coord] == 0 || rebote )
	{
		//cambias el valor a las siguientes posiciones
		agent[y*agent_width+x].x = agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z) * agent[y*agent_width+x].w) ;
		agent[y*agent_width+x].y = agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z) * agent[y*agent_width+x].w) ;
		//pones el id del agente en la casilla del mundo que esta ocupando
		d_world[coord] = ids[y*agent_width+x].x ;
		//liberas la casilla ocupada anteriormente
		d_world[coord_old] = 0;
	}
	else{
		cx = round( agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + PI/4)  *  agent[y*agent_width+x].w ) ) ;
		cy = (  round( agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + PI/4)  * agent[y*agent_width+x].w) ) -1  ) * world_width ;
		coord =  cy + cx ;
		if( d_world[coord] == 0)
		{
			agent[y*agent_width+x].x = agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + PI/4) *agent[y*agent_width+x].w) ;
			agent[y*agent_width+x].y = agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + PI/4) *agent[y*agent_width+x].w) ;
			d_world[coord] = ids[y*agent_width+x].x ;
			d_world[coord_old] = 0;
		}
		else{
			cx = round( agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + 2*PI/4)  * agent[y*agent_width+x].w) ) ;
			cy = ( round( agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + 2*PI/4)  * agent[y*agent_width+x].w) ) - 1) * world_width;
			coord =  cy + cx ;
			if( d_world[coord] == 0)
			{
				agent[y*agent_width+x].x = agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + 2*PI/4) *agent[y*agent_width+x].w) ;
				agent[y*agent_width+x].y = agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + 2*PI/4) *agent[y*agent_width+x].w) ;
				d_world[coord] = ids[y*agent_width+x].x ;
				d_world[coord_old] = 0;
			}
			else{
				cx = round( agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + 3*PI/4)  * agent[y*agent_width+x].w) ) ;
				cy = ( round( agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + 3*PI/4) * agent[y*agent_width+x].w) ) -1 ) * world_width ;
				coord =  cy + cx ;
				if( d_world[coord] == 0)
				{
					agent[y*agent_width+x].x = agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + 3*PI/4) *agent[y*agent_width+x].w) ;
					agent[y*agent_width+x].y = agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + 3*PI/4) *agent[y*agent_width+x].w) ;
					d_world[coord] = ids[y*agent_width+x].x ;
					d_world[coord_old] = 0;
				}
				else{
					cx = round( agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + PI)  * agent[y*agent_width+x].w) ) ;
					cy = ( round( agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + PI) * agent[y*agent_width+x].w) ) -1 ) * world_width;
					coord =  cy + cx ;
					if( d_world[coord] == 0)
					{
						agent[y*agent_width+x].x = agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + PI) *agent[y*agent_width+x].w) ;
						agent[y*agent_width+x].y = agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + PI) *agent[y*agent_width+x].w) ;
						d_world[coord] = ids[y*agent_width+x].x ;
						d_world[coord_old] = 0;
					}
					else{
						cx = round( agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + 5*PI/4)  * agent[y*agent_width+x].w) ) ;
						cy = ( round( agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + 5*PI/4)  * agent[y*agent_width+x].w) ) - 1 ) * world_width;
						coord =  cy + cx ;
						if( d_world[coord] == 0)
						{
							agent[y*agent_width+x].x = agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + 5*PI/4) *agent[y*agent_width+x].w) ;
							agent[y*agent_width+x].y = agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + 5*PI/4) *agent[y*agent_width+x].w) ;
							d_world[coord] = ids[y*agent_width+x].x ;
							d_world[coord_old] = 0;
						}
						else{
							cx = round( agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + 6*PI/4) * agent[y*agent_width+x].w) ) ;
							cy = ( round( agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + 6*PI/4) * agent[y*agent_width+x].w) ) -1 ) * world_width;
							coord =  cy + cx ;
							if( d_world[coord] == 0)
							{
								agent[y*agent_width+x].x = agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + 6*PI/4) *agent[y*agent_width+x].w) ;
								agent[y*agent_width+x].y = agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + 6*PI/4) *agent[y*agent_width+x].w) ;
								d_world[coord] = ids[y*agent_width+x].x ;
								d_world[coord_old] = 0;
							}
							else{
								cx = round( agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + 7*PI/4) * agent[y*agent_width+x].w) ) ;
								cy = ( round( agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + 7*PI/4) * agent[y*agent_width+x].w) ) - 1 ) * world_width;
								coord =  cy + cx ;
								if( d_world[coord] == 0)
								{
									agent[y*agent_width+x].x = agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z + 7*PI/4) *agent[y*agent_width+x].w) ;
									agent[y*agent_width+x].y = agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z + 7*PI/4) *agent[y*agent_width+x].w) ;
									d_world[coord] = ids[y*agent_width+x].x ;
									d_world[coord_old] = 0;
								}
								else{
									//si todas las posiciones a su alrededor estan ocupadas se queda donde esta y marcas
									//ocupada de nuevo esa posicion
									//d_world[coord_old] = 1;
	//si todas las posiciones a su alrededor estan ocupadas avanzas en la direccion original aunque se colisione
	cx = round( agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z) * agent[y*agent_width+x].w) ) ;
	cy = ( round( agent[y*agent_width+x].y +  ( sin(agent[y*agent_width+x].z)  * agent[y*agent_width+x].w) ) - 1 )* world_width ;
	coord =  cx + cy ;

	agent[y*agent_width+x].x = agent[y*agent_width+x].x + ( cos(agent[y*agent_width+x].z) *agent[y*agent_width+x].w) ;
	agent[y*agent_width+x].y = agent[y*agent_width+x].y + ( sin(agent[y*agent_width+x].z) *agent[y*agent_width+x].w) ;
	d_world[coord] = ids[y*agent_width+x].x ;
	d_world[coord_old] = 0;
								}
							} //7*PI/4
						} //6*PI/4
					}//5*PI/4
				}//PI
			}//3*PI/4
		 }//PI/2
	}//PI/4

	return coord;

}

#endif

__device__ void d_interchange(unsigned int x, unsigned int y, float4 *agent, float4 *ids, int *d_world, int agent_width, int world_height_node,
		int world_width_node, int num_comp_nodes, int coord)
{
   int lookup_node4[2][2];
   lookup_node4[0][0] = 1;
   lookup_node4[0][1] = 2;
   lookup_node4[1][0] = 3;
   lookup_node4[1][1] = 4;

   int lookup_node16[4][4];
   lookup_node16[0][0] = 1;
   lookup_node16[0][1] = 2;
   lookup_node16[0][2] = 3;
   lookup_node16[0][3] = 4;
   lookup_node16[1][0] = 5;
   lookup_node16[1][1] = 6;
   lookup_node16[1][2] = 7;
   lookup_node16[1][3] = 8;
   lookup_node16[2][0] = 9;
   lookup_node16[2][1] = 10;
   lookup_node16[2][2] = 11;
   lookup_node16[2][3] = 12;
   lookup_node16[3][0] = 13;
   lookup_node16[3][1] = 14;
   lookup_node16[3][2] = 15;
   lookup_node16[3][3] = 16;

   int lookup_node9[3][3];
   lookup_node9[0][0] = 1;
   lookup_node9[0][1] = 2;
   lookup_node9[0][2] = 3;
   lookup_node9[1][0] = 4;
   lookup_node9[1][1] = 5;
   lookup_node9[1][2] = 6;
   lookup_node9[2][0] = 7;
   lookup_node9[2][1] = 8;
   lookup_node9[2][2] = 9;

	if (num_comp_nodes == 4)
	{
		ids[y*agent_width+x].z = lookup_node4[(int)agent[y*agent_width+x].y/world_height_node][(int)agent[y*agent_width+x].x/world_width_node];
		d_world[coord] = 0 ;
	}
	else if (num_comp_nodes == 16)
	{
		ids[y*agent_width+x].z = lookup_node16[(int)agent[y*agent_width+x].y/world_height_node][(int)agent[y*agent_width+x].x/world_width_node];
		d_world[coord] = 0 ;
	}
	else if (num_comp_nodes == 9)
	{
		ids[y*agent_width+x].z = lookup_node9[(int)agent[y*agent_width+x].y/world_height_node][(int)agent[y*agent_width+x].x/world_width_node];
		d_world[coord] = 0 ;
	}

}

__device__ bool d_border(int x, int y, float4 *agent, int world_width, int agent_width, int world_height_node,
		int world_width_node, int pid, int num_comp_nodes)
{
	int border_width = 5;
	int nodes_per_row = sqrtf(num_comp_nodes);
	int x_limit_right = (( ( (pid - 1) % nodes_per_row ) + 1 ) * world_width_node);
	int x_limit_left = (( (pid - 1) % nodes_per_row )  * world_width_node);
	int num_row = (pid - 1) / nodes_per_row;
	int y_limit_up = (( num_row + 1 ) * world_height_node);
	int y_limit_down = (num_row  * world_height_node);

	//getting coords
	int cx = round( agent[y*agent_width+x].x ) ;
	int cy = round( agent[y*agent_width+x].y ) ;


	if( cx < x_limit_left + border_width )
	{
		return true;
	}

	if( cx > x_limit_right - border_width )
	{
		return true;
	}

	if( cy < y_limit_down + border_width )
	{
		return true;
	}

	if( cy > y_limit_up - border_width )
	{
		return true;
	}

	return false;
}

///////////////////////////////////////////////////////////////////////////////
//! Kernel to modify internal agents positions
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void kernel_internal(float4 *agent, float4 *ids, int *d_world, int world_width, int world_height, int world_height_node,
		int world_width_node, int pid, int num_comp_nodes, int count_agents_total, int grid_width)
{
   unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
   unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
   int coord;
   int agent_width = grid_width;



   if ( y * agent_width + x < count_agents_total ) //verify bounds
   {
	   if(ids[y*agent_width+x].z == pid) //check if agent is active
	   {
		   if(!d_border(x, y, agent, world_width, agent_width, world_height_node, world_width_node, pid, num_comp_nodes)) //check is not in the border
		   {
			   coord = d_behavior(x, y, agent, ids, d_world, world_width, world_height, agent_width);
			   //d_interchange(x, y, agent, ids, d_world, agent_width, world_height_node, world_width_node, num_comp_nodes, coord);
		   }
	   }
	   else
		   ids[y*agent_width+x].z = -1;
   }
}

///////////////////////////////////////////////////////////////////////////////
//! Kernel to modify border agents positions
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void kernel_border(float4 *agent, float4 *ids, int *d_world, int world_width, int world_height, int world_height_node,
		int world_width_node, int pid, int num_comp_nodes, int count_agents_total, float4 *agent_fr, float4 *ids_fr, int grid_width)
{
   unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
   unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
   int agent_width = grid_width;
   int coord;
   /*
   if ( x == 0 )
 	   printf("kernel_border \n");
*/
   if ( y * agent_width + x < count_agents_total ) //verify bounds of the array
   {
	   if(ids[y*agent_width+x].z == pid) //check if agent is active
	   {
		   ids[y*agent_width+x].w += 1; //iterations counter

		   if(d_border(x, y, agent, world_width, agent_width, world_height_node, world_width_node, pid, num_comp_nodes)) //check is in the border
		   {
			   coord = d_behavior(x, y, agent, ids, d_world, world_width, world_height, agent_width);
			   d_interchange(x, y, agent, ids, d_world, agent_width, world_height_node, world_width_node, num_comp_nodes, coord);
			   agent_fr[y*agent_width+x] = agent[y*agent_width+x];
			   ids_fr[y*agent_width+x] = ids[y*agent_width+x];
		   }
	   }
	   else
		   ids[y*agent_width+x].z = -1;
   }
}

///////////////////////////////////////////////////////////////////////////////
//! Kernel to load positions to draw throught OpenGL
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void kernel_draw(float4 *d_agent, float4 *d_tmp, float4 *ids, int agent_width, int count_agents_total, int pid)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if ( y * agent_width + x < count_agents_total)  //verify bounds of the array
	{
		if (ids[y*agent_width+x].z == pid)
		{
		d_tmp[y*agent_width+x].x = d_agent[y*agent_width+x].x;
		d_tmp[y*agent_width+x].y = d_agent[y*agent_width+x].y;
		d_tmp[y*agent_width+x].z = 0;
		d_tmp[y*agent_width+x].w = 1;
		}
		else
		{
		d_tmp[y*agent_width+x].x = -1;
		d_tmp[y*agent_width+x].y = -1;
		d_tmp[y*agent_width+x].z = 0;
		d_tmp[y*agent_width+x].w = 1;
		}

	}
}

// CUDA computation on each node
// No MPI here, only CUDA
extern "C" void launch_kernel_border(float4 *d_agents_in, float4 *d_agents_ids, int *d_world, int world_width, int world_height, int world_height_node,
		int world_width_node, int pid, int num_comp_nodes, int count_agents_total, float4 *d_agents_in_fr, float4 *d_agents_ids_fr,	hipStream_t stream0)
{
    // execute the kernel
    int block_width = 8;
    int block_height = 8;
    dim3 block(block_width, block_height, 1);
    int agent_width = sqrt(count_agents_total);
    dim3 grid((agent_width / block.x) + 1, (agent_width / block.y) + 1, 1);

    kernel_border<<< grid, block>>>(d_agents_in, d_agents_ids, d_world, world_width, world_height, world_height_node, world_width_node,
    		pid, num_comp_nodes, count_agents_total, d_agents_in_fr, d_agents_ids_fr, agent_width);
}

extern "C" void launch_kernel_internal(float4 *d_agents_in, float4 *d_agents_ids, int *d_world, int world_width, int world_height,
		int world_height_node, int world_width_node, int pid, int num_comp_nodes, int count_agents_total)
{
    // execute the kernel
    int block_width = 8;
    int block_height = 8;
    dim3 block(block_width, block_height, 1);
    int agent_width = sqrt(count_agents_total);
    dim3 grid((agent_width / block.x) + 1, (agent_width / block.y) + 1, 1);

    kernel_internal<<< grid, block>>>(d_agents_in, d_agents_ids, d_world, world_width, world_height, world_height_node, world_width_node,
    		pid, num_comp_nodes, count_agents_total, agent_width);
}

extern "C" void launch_kernel_draw(float4 *d_agents_in, float4 *d_tmp, float4 *d_ids, int count_agents_total, int pid)
{
    // execute the kernel
    int block_width = 8;
    int block_height = 8;
    dim3 block(block_width, block_height, 1);
    int agent_width = sqrt(count_agents_total);
    dim3 grid((agent_width / block.x) + 1, (agent_width / block.y) + 1, 1);

    kernel_draw<<< grid, block>>>(d_agents_in, d_tmp, d_ids, agent_width, count_agents_total, pid);
}

#endif




