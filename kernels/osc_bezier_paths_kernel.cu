#ifndef _PATHS_KERNEL_H_
#define _PATHS_KERNEL_H_


#define DEG2RAD	0.01745329251994329576f
#define RAD2DEG 57.29577951308232087679f

//#define HDF5Server


//
//=======================================================================================
//
//
//=======================================================================================
//
__global__ void kernel2(	float*			npos,
							float*			ppos,
							float*			cpos,
							unsigned int	width, 
							unsigned int	height,
							float			plane_scale,
							float			time 		)
{
    unsigned int	x				= blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int	y				= blockIdx.y * blockDim.y + threadIdx.y;
	//unsigned int	index			= y * width + x;
	unsigned int	index			= 4 * ( x + y * blockDim.x * gridDim.x );

#ifdef HDF5Server
	if( npos[index+0] == -plane_scale/2.0f && npos[index+2] == -plane_scale/2.0f )			//GOING TOWARDS 0,0
	{
		cpos[index+0]					= -plane_scale * 10.0f;
		cpos[index+2]					= -plane_scale * 10.0f;
	}
	else if( ppos[index+0] == -plane_scale/2.0f && ppos[index+2] == -plane_scale/2.0f )		//COMING FROM 0,0
	{
		cpos[index+0]					= npos[index+0];
		cpos[index+2]					= npos[index+1];
	}
	else
	{
#endif
		float3	tpos;
		tpos.x					= ppos[index+0] + time*(npos[index+0]-ppos[index+0]);
		tpos.y					= 0.0f;
		tpos.z					= ppos[index+2] + time*(npos[index+2]-ppos[index+2]);
#ifdef HDF5Server
		//if( tpos.x == cpos[index+0] && tpos.z == cpos[index+2] )							//NO MOTION
		//if( ppos[index+0] == npos[index+0] && ppos[index+2] == npos[index+2] )							//NO MOTION
//		{
//			cpos[index+0]				= -plane_scale * 10.0f;
//			cpos[index+2]				= -plane_scale * 10.0f;
//		}
//		else
//		{
#endif
			cpos[index+0]				= tpos.x;
			cpos[index+1]				= tpos.y;
			cpos[index+2]				= tpos.z;
			cpos[index+3]				= atan2( npos[index+0]-ppos[index+0], npos[index+2]-ppos[index+2] );
#ifdef HDF5Server
		}
	}
#endif
}
//
//=======================================================================================
//


__global__ void mpi_on_node_kernel(	float*			npos,
									float*			ppos,
									float*			cpos,
									float4*			node_agents_pos,
									float4*			node_agents_ids,
									float			plane_scale,
									int				pid,
									float			time					)
{
	unsigned int	x				= blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int	y				= blockIdx.y * blockDim.y + threadIdx.y;
	//unsigned int	index			= y * width + x;
	unsigned int	index			= 4 * ( x + y * blockDim.x * gridDim.x );

	ppos[index+0] = npos[index+0];
	ppos[index+1] = npos[index+1];
	ppos[index+2] = npos[index+2];
	ppos[index+3] = npos[index+3];



	if ( node_agents_ids[index].z == pid )
	{
		npos[index+0] = node_agents_pos[index].x / 4096.0f  * plane_scale - plane_scale/2.0f ;
		npos[index+1] = 0.0f;
		npos[index+2] = node_agents_pos[index].y / 4096.0f  * plane_scale - plane_scale/2.0f ;
		npos[index+3] = 0.0f;
	}

	else
	{
		npos[index+0] = -plane_scale * 10.0f;
		npos[index+1] = -plane_scale * 10.0f;
		npos[index+2] = -plane_scale * 10.0f;
		npos[index+3] = 1.0f;
	}

	float3	tpos;
	tpos.x					= ppos[index+0] + time*(npos[index+0]-ppos[index+0]);
	tpos.y					= 0.0f;
	tpos.z					= ppos[index+2] + time*(npos[index+2]-ppos[index+2]);

	cpos[index+0]			= tpos.x;
	cpos[index+1]			= tpos.y;
	cpos[index+2]			= tpos.z;
	cpos[index+3]			= atan2( npos[index+0]-ppos[index+0], npos[index+2]-ppos[index+2] );



}
//
//=======================================================================================
//
/*
__global__ void mpi_on_node_kernel_passthru (	float4*	d_agent,
												float*	cpos,
												int		agent_width,
												int		count_agents_total,
												int		)
												*/
//
//=======================================================================================
//
extern "C" void launch_osc_bezier_kernel (	float*			npos,
											float*			ppos,
											float*			cpos,
											unsigned int	mesh_width,
											unsigned int	mesh_height,
											float			plane_scale,
											float			time		)
{
    dim3 block( 4, 4, 1 );
    dim3 grid( mesh_width / block.x, mesh_height / block.y, 1 );
    kernel2<<<grid, block>>>( npos, ppos, cpos, mesh_width, mesh_height, plane_scale, time );
}
//
//=======================================================================================
//
extern "C" void launch_mpi_on_nodes_kernel (	float*			npos,
												float*			ppos,
												float*			cpos,
												float4*			node_agents_pos, // position calculated on slaves nodes (compute_process, behaviorMPI.cpp)
												float4*			node_agents_ids,
												unsigned int 	crowd_width,
												unsigned int 	crowd_height,
												float 			plane_scale,
												int				pid,
												float			time		)
{
    dim3 block( 4, 4, 1 );
    dim3 grid( crowd_width / block.x, crowd_height / block.y, 1 );
    mpi_on_node_kernel<<<grid, block>>>( npos, ppos, cpos, node_agents_pos, node_agents_ids, plane_scale, pid, time );
}
//
//=======================================================================================
//
/*
extern "C" void launch_mpi_on_nodes_kernel_passthru( float4* 		d_agents,
													float*  		cpos,
													int				count_agents_total,
													int 			pid 				)
{
    // execute the kernel
    int block_width = 8;
    int block_height = 8;
    dim3 block(block_width, block_height, 1);
    int agent_width = sqrt(count_agents_total);
    dim3 grid((agent_width / block.x) + 1, (agent_width / block.y) + 1, 1);

    mpi_on_node_kernel_passthru<<< grid, block>>>(d_agents_in, cpos, agent_width, count_agents_total, pid);


}
*/
//
//=======================================================================================
//

#endif
